#include "hip/hip_runtime.h"
// Timothy Fischer

#include <iostream>

#include ""
#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

#define N 1024
#define SIZE 1000
#define MAX_ITER 100
#define MAX_TEMP_ERROR 0.01

double Temperature[N][N];
double Temperature_last[N][N];

double* d_Temperature;
double* d_Temperature_last;

void init()
{
    int i,j;

    // set cnter grid 
    for(i = 0; i <= SIZE+1; i++){
        for (j = 0; j <= SIZE+1; j++){
            Temperature_last[i+1][j+1] = 0.0;
        }
    }

    // set left side to 0 and right to a linear increase
    for(i = 0; i <= SIZE+1; i++) {
        Temperature_last[i][0] = 0.0;
        Temperature_last[i][SIZE+1] = (100.0/SIZE)*i;
    }
    
    // set top to 0 and bottom to linear increase
    for(j = 0; j <= SIZE+1; j++) {
        Temperature_last[0][j] = 0.0;
        Temperature_last[SIZE+1][j] = (100.0/SIZE)*j;
    }
}

__device__ double my_fmax(double a, double b) 
{
  return (a > b) ? a : b;
}

__global__ void updateTemp(double** Temperature, double** Temperature_last, int max_iter, double max_error)
{
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);

    int iter = 0;
    int dt = 100;

    int i = idx%1024;
    int j = (int)idx/1024;

    while(iter < max_iter && dt > max_error)
    {
        if (i < SIZE+2 && j < SIZE+2)
        {
            Temperature[i][j] = 0.25 * (Temperature_last[i+1][j] + Temperature_last[i-1][j] + Temperature_last[i][j+1] + Temperature_last[i][j-1]);

            iter++;

            dt = my_fmax( fabs(Temperature[i][j]-Temperature_last[i][j]), dt);
            Temperature_last[i][j] = Temperature[i][j];
        }
    }
}


int main(int argc, char** argv)
{ 
    init();

    // initialise CUDA timing
	float milli;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    int max_iter = MAX_ITER;
    double max_error = MAX_TEMP_ERROR;
    int size = N * N * sizeof(double);

    // Allocate device memory
    checkCudaErrors(hipMalloc((void**)&d_Temperature, size));
    checkCudaErrors(hipMalloc((void**)&d_Temperature_last, size));

    // Copy initialized matrix from host to device
    checkCudaErrors(hipMemcpy(d_Temperature, Temperature, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_Temperature_last, Temperature_last, size, hipMemcpyHostToDevice));

    // Set Kernel Parameters
    dim3 matrixBlock(32, 32, 1);
    dim3 matrixGrid(32, 32, 1);

    // Launch Kernel
    hipEventRecord(start); 
    checkCudaErrors(hipDeviceSynchronize());
    updateTemp <<< matrixBlock, matrixGrid >>> (&d_Temperature, &d_Temperature_last, max_iter, max_error);
    hipEventRecord(stop);
	checkCudaErrors(hipEventSynchronize(stop));
	hipEventElapsedTime(&milli, start, stop);  

    printf("updateTemp <<<(%d,%d), (%d,%d)>>> (ms): %f \n", matrixGrid.x, matrixGrid.y,
        matrixBlock.x, matrixBlock.y, milli);

    checkCudaErrors(hipFree(Temperature));
    checkCudaErrors(hipFree(Temperature_last));
    


    return 0;
}
